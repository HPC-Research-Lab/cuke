#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <torch/torch.h>
using namespace std;

#define BLOCK_SIZE 256
#define C 16
#define T 3
#define DIV(x, ts) ((x) % (ts) != 0 ? (x) / (ts) + 1 : (x) / (ts))

__device__ void swap(int& a, int& b, int& a_idx, int& b_idx) {
    int tmp = a;
    a = b;
    b = tmp;
    tmp = a_idx;
    a_idx = b_idx;
    b_idx = tmp;
}

__device__ void bitonic_sort(int* arr, int* ord) {
    __shared__ int shared_arr[C];
    __shared__ int shared_ord[C];

    int tid = threadIdx.x;
    shared_arr[tid] = arr[tid];
    shared_ord[tid] = tid;
    __syncthreads();

    for (int k = 2; k <= C; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            __syncthreads();
            int ixj = tid ^ j;
            if (ixj > tid) {
                if ((tid & k) == 0 && shared_arr[tid] > shared_arr[ixj])
                    swap(shared_arr[tid], shared_arr[ixj], shared_ord[tid], shared_ord[ixj]);
                if ((tid & k) != 0 && shared_arr[tid] < shared_arr[ixj])
                    swap(shared_arr[tid], shared_arr[ixj], shared_ord[tid], shared_ord[ixj]);
            }
        }
    }

    __syncthreads();
    arr[tid] = shared_arr[tid];
    ord[shared_ord[tid]] = tid;
}

__global__ void build_index(torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> indices, torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> r_Uniq, torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> r_Buffer, torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> uniq_cnt, int rel_num){
    __shared__ int idx[C], ord[C], ibuf[C], iuniq[C], pcount[C], ord_uniq[C];
    int tid = threadIdx.x;
    idx[tid] = indices[blockIdx.x * C + tid];
    ord[tid] = tid;
    ord_uniq[tid] = tid;
    pcount[tid] = 0;
    __syncthreads();

    bitonic_sort(idx, ord);
    ibuf[tid] = (tid > 0 && idx[tid] > idx[tid-1]) ? 1:0;
    __syncthreads();
    
    for (int offset = 1; offset < C; offset *= 2) {
        __syncthreads();
        if (tid >= offset) {
            ibuf[tid] += ibuf[tid - offset];
        }
    }
    
    if (tid == 0) { pcount[ibuf[C-1]+1] = C; }
    else if (idx[tid] > idx[tid-1]) {
            pcount[ibuf[tid]] = tid; }
    iuniq[tid] = rel_num;
    __syncthreads();

    // exceed threshold
    if (tid > 0 && pcount[tid]-pcount[tid-1]>T) {
        iuniq[tid-1] = idx[pcount[tid]-1]; }
    __syncthreads();

    bitonic_sort(iuniq, ord_uniq);

    int temp = ord_uniq[ibuf[tid]];
    if (iuniq[temp] < rel_num){
        ibuf[tid] = temp;
    }else{
        ibuf[tid] = idx[tid] + C;
    }
    if (iuniq[tid] < rel_num && iuniq[tid+1] == rel_num){
        uniq_cnt[blockIdx.x] = tid+1;
    }
    r_Buffer[blockIdx.x][tid] = ibuf[ord[tid]];
    r_Uniq[blockIdx.x][tid] = iuniq[tid];
}


void gpu_sort(torch::Tensor head, torch::Tensor tail, torch::Tensor relation, torch::Tensor r_Uniq, torch::Tensor r_Buffer, torch::Tensor uniq_cnt, int batch, int group_size, int rel_num) {
    // int batch=4096;
    dim3 nblocks(DIV(batch, C));
    dim3 nthreads(32, C);

    torch::Tensor sorted_indices = torch::argsort(relation);
    std::cout << "Original Head: " << relation << std::endl;

    head = torch::index_select(head, 0, sorted_indices);
    tail = torch::index_select(tail, 0, sorted_indices);
    relation = torch::index_select(relation, 0, sorted_indices);

    
    std::cout << "Sorted Head: " << relation << std::endl;

    build_index<<< batch/C, C>>>(relation.packed_accessor32<int, 1, torch::RestrictPtrTraits>(), r_Uniq.packed_accessor32<int, 2, torch::RestrictPtrTraits>(), r_Buffer.packed_accessor32<int, 2, torch::RestrictPtrTraits>(), uniq_cnt.packed_accessor32<int, 1, torch::RestrictPtrTraits>(), rel_num);

    std::cout << "uniq: " << r_Uniq << std::endl;
    std::cout << "buffer: " << r_Buffer << std::endl;
    std::cout << "uniq_cnt: " << uniq_cnt << std::endl;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("gpu_sort", &gpu_sort, "ss");
}