#include "hip/hip_runtime.h"
#include <stdio.h>
#include<iostream>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
using namespace std;

#define BLOCK_SIZE 256
#define C 16
#define T 2


__device__ void swap(long& a, long& b, long& a_idx, long& b_idx) {
    int tmp = a;
    a = b;
    b = tmp;
    tmp = a_idx;
    a_idx = b_idx;
    b_idx = tmp;
}

__device__ void bitonic_sort(long* arr, long* ord) {
    __shared__ long shared_arr[C];
    __shared__ long shared_ord[C];

    int tid = threadIdx.x;
    shared_arr[tid] = arr[tid];
    shared_ord[tid] = tid;
    __syncthreads();

    for (int k = 2; k <= C; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            __syncthreads();
            int ixj = tid ^ j;
            if (ixj > tid) {
                if ((tid & k) == 0 && shared_arr[tid] > shared_arr[ixj])
                    swap(shared_arr[tid], shared_arr[ixj], shared_ord[tid], shared_ord[ixj]);
                if ((tid & k) != 0 && shared_arr[tid] < shared_arr[ixj])
                    swap(shared_arr[tid], shared_arr[ixj], shared_ord[tid], shared_ord[ixj]);
            }
        }
    }

    __syncthreads();
    arr[tid] = shared_arr[tid];
    ord[shared_ord[tid]] = tid;
}

__global__ void build_index(long * indices, long * uniq_idx, long * buf_idx, long * uniq_cnt){
    __shared__ long idx[C], ord[C], ibuf[C], iuniq[C], count[C], ord_uniq[C];
    int tid = threadIdx.x;
    idx[tid] = indices[blockIdx.x * C + tid];
    ord[tid] = tid;
    ord_uniq[tid] = tid;
    count[tid] = 0;
    __syncthreads();

    bitonic_sort(idx, ord);
    ibuf[tid] = (tid > 0 && idx[tid] > idx[tid-1]) ? 1:0;
    __syncthreads();
    
    for (int offset = 1; offset < C; offset *= 2) {
        __syncthreads();
        if (tid >= offset) {
            ibuf[tid] += ibuf[tid - offset];
        }
    }
    
    // buf_idx[blockIdx.x * C + tid] = ibuf[ord[tid]];
    if (tid == 0) { count[ibuf[C-1]+1] = C; }
    else if (idx[tid] > idx[tid-1]) {
            count[ibuf[tid]] = tid; }
    iuniq[tid] = 999;
    __syncthreads();

    // exceed threshold
    if (tid > 0 && count[tid]-count[tid-1]>T) {
        iuniq[tid-1] = idx[count[tid]-1]; }
    __syncthreads();

    bitonic_sort(iuniq, ord_uniq);

    int temp = ord_uniq[ibuf[tid]];
    // if(threadIdx.x == 0 && blockIdx.x == 0){
    //     for(int i=0;i<C;i++){
    //         printf("%d ", iuniq[i]);
    //     }
    // }
    if (iuniq[temp] < 999){
        ibuf[tid] = temp;
    }else{
        ibuf[tid] = idx[tid] + C;
    }
    if (iuniq[tid] < 999 && iuniq[tid+1] == 999){
        uniq_cnt[blockIdx.x] = tid+1;
    }
    buf_idx[blockIdx.x * C + tid] = ibuf[ord[tid]];
    uniq_idx[blockIdx.x * C + tid] = iuniq[tid];
}


int main() {
    // long data[64] = { 2, 5, 5, 3, 1, 2, 5, 3, 1, 3, 5, 5, 3, 4, 5, 3, 5, 4, 2, 2, 5, 5, 1, 3, 5, 5, 3, 1, 4, 5, 5, 2, 1, 5, 2, 5, 7, 5, 5, 4, 5, 4, 6, 6, 4, 3, 4, 2, 2, 2, 3, 4, 7, 2, 1, 3, 1, 3, 1, 1, 4, 5, 2, 1};
    // long data[256] = {
    //     42, 41, 42, 42, 42, 42, 30, 50, 20,  1, 43, 18,  0, 17, 42, 42, 
    //     42, 40, 14, 44, 19, 41, 41, 42, 23, 17,  8, 46, 42, 43, 46, 19, 
    //     38, 43, 17, 50, 42, 42, 42, 46, 43, 42, 43, 45, 33, 43, 43, 42, 
    //      6, 49,  5, 13, 43, 42, 45, 43,  3, 43, 42, 42, 42, 43, 43, 45, 
    //     41, 36, 39, 50, 41, 42, 43, 45, 10, 44, 42, 19, 42, 42, 45, 42, 
    //     42,  6, 42, 14, 46, 19, 42,  0, 22, 42, 45, 40, 43, 43,  0, 43, 
    //     45, 45, 42, 38, 13, 45, 42, 30, 45, 46, 50, 46, 43, 45, 43, 42,
    //     42, 43, 42, 43, 50, 41, 42, 50, 43, 43, 42, 23, 42, 46, 50, 50, 
    //     43, 42, 34, 50, 22, 43, 46, 42, 42, 42, 46, 43, 42, 42, 45, 42, 
    //     42, 43, 43, 43, 31, 38, 43, 13, 43, 43, 43, 42,  4,  0, 50, 18, 
    //     50, 43, 43, 45, 42, 46, 44, 42, 48, 42, 42, 43, 22, 42, 42, 45, 
    //     42, 43, 42, 43, 42, 45, 22, 30, 42, 41, 50, 41,  2, 45, 42, 42, 
    //     42, 43,  5, 43, 43, 50, 38, 42, 11, 46, 43, 43,  8, 50, 45, 48, 
    //     40, 42, 42, 50, 46, 42, 42, 22, 46, 46, 50,  6, 14, 42, 11, 43,
    //     42, 35, 42, 43, 45, 13, 42, 42, 43, 42, 43, 45, 42, 42, 41, 43, 
    //     42, 14, 31, 42,  2, 42, 38, 43, 42,  2, 42, 19, 31, 42, 40, 43
    // };
    long data[512] = {0,  0,  0,  0,  0,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  3,  3,  3,  4,  4,  4,  5,  5,  5,  5,
         5,  5,  5,  5,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  7,  8,  8,  8, 10, 10, 10, 10,
        11, 11, 11, 11, 11, 11, 11, 12, 12, 12, 12, 13, 13, 13, 13, 13, 13, 13, 13, 14, 14, 14, 14, 14, 14, 14, 14, 14,
        18, 18, 18, 18, 18, 19, 19, 19, 19, 19, 19, 19, 22, 22, 22, 22, 22, 22, 22, 22, 22, 22, 23, 23, 23, 23, 23, 28,
        30, 31, 31, 31, 31, 31, 31, 31, 31, 31, 33, 33, 33, 33, 35, 35, 35, 36, 37, 37, 38, 38, 38, 38, 38, 39, 39, 39,
        39, 39, 39, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 41, 41, 41, 41, 41, 41, 41, 41, 41, 41, 41,
        41, 41, 41, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42,
        42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42,
        42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42,
        42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42,
        42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42, 42,
        42, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43,
        43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43,
        43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43, 43,
        44, 44, 44, 44, 44, 44, 44, 44, 44, 44, 44, 44, 44, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45,
        45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46,
        46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 46, 47, 48, 48, 48, 48, 50, 50, 50, 50, 50, 50,
        50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
        50, 50, 50, 50, 50, 50, 50, 50};
    int n = sizeof(data) / sizeof(data[0]);
    printf("%d\n", n);
    long  *uniq_idx=(long*)malloc(sizeof(long) * n), *buf_idx=(long*)malloc(sizeof(long) * n);
    

    long *d_data, *d_uniq_idx, *d_buf_idx, *d_uniq_cnt, *uniq_cnt=(long*)malloc(sizeof(long) * n/C);
    hipMalloc(&d_data, n * sizeof(long));
    hipMalloc(&d_uniq_idx, n * sizeof(long));
    hipMalloc(&d_buf_idx, n * sizeof(long));

    hipMalloc(&d_uniq_cnt, n * sizeof(long));

    hipMemcpy(d_data, data, n * sizeof(long), hipMemcpyHostToDevice);


    build_index<<< n/C, C>>>(d_data, d_uniq_idx, d_buf_idx, d_uniq_cnt);

    // print result
    // for (int i = 0; i < n; i++) {
    //     printf("%d ", data[i]);
    // }
    // printf("\n%d\n", n);

    hipMemcpy(uniq_idx, d_uniq_idx, n*sizeof(long), hipMemcpyDeviceToHost);
    hipMemcpy(buf_idx, d_buf_idx, n*sizeof(long), hipMemcpyDeviceToHost);
    hipMemcpy(uniq_cnt, d_uniq_cnt, n/C*sizeof(long), hipMemcpyDeviceToHost);
    printf("unique idx:\n");
    for(int i=0;i<n;++i){
        if(i%C == 0){
            printf("\n");
        }
        // printf("%d,%d ", uniq_idx[i], buf_idx[i]);
        printf("%ld ", uniq_idx[i]);
    }
    printf("buffer idx:\n");
    for(int i=0;i<n;++i){
        if(i%C == 0){
            printf("\n");
        }
        // printf("%d,%d ", uniq_idx[i], buf_idx[i]);
        printf("%ld ", buf_idx[i]);
    }
    printf("\n");
    for(int i=0;i<n/C;++i){
        printf("%ld ", uniq_cnt[i]);
    }
    return 0;
}