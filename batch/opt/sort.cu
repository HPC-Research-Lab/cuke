#include "hip/hip_runtime.h"
__device__ void swap(long& a, long& b, long& a_idx, long& b_idx) {
    int tmp = a;
    a = b;
    b = tmp;
    tmp = a_idx;
    a_idx = b_idx;
    b_idx = tmp;
}

__device__ void bitonic_sort(long* arr, long* ord) {
    __shared__ long shared_arr[C];
    __shared__ long shared_ord[C];

    int tid = threadIdx.x;
    shared_arr[tid] = arr[tid];
    shared_ord[tid] = tid;
    __syncthreads();

    for (int k = 2; k <= C; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            __syncthreads();
            int ixj = tid ^ j;
            if (ixj > tid) {
                if ((tid & k) == 0 && shared_arr[tid] > shared_arr[ixj])
                    swap(shared_arr[tid], shared_arr[ixj], shared_ord[tid], shared_ord[ixj]);
                if ((tid & k) != 0 && shared_arr[tid] < shared_arr[ixj])
                    swap(shared_arr[tid], shared_arr[ixj], shared_ord[tid], shared_ord[ixj]);
            }
        }
    }

    __syncthreads();
    arr[tid] = shared_arr[tid];
    ord[shared_ord[tid]] = tid;
}

__global__ void build_index(long * indices, long * uniq_idx, long * buf_idx, int * uniq_cnt){
    __shared__ long idx[C], ord[C], ibuf[C], iuniq[C], count[C], ord_uniq[C];
    int tid = threadIdx.x;
    idx[tid] = indices[blockIdx.x * C + tid];
    ord[tid] = tid;
    ord_uniq[tid] = tid;
    count[tid] = 0;
    __syncthreads();

    bitonic_sort(idx, ord);
    ibuf[tid] = (tid > 0 && idx[tid] > idx[tid-1]) ? 1:0;
    __syncthreads();
    
    for (int offset = 1; offset < C; offset *= 2) {
        __syncthreads();
        if (tid >= offset) {
            ibuf[tid] += ibuf[tid - offset];
        }
    }
    
    if (tid == 0) { count[ibuf[C-1]+1] = C; }
    else if (idx[tid] > idx[tid-1]) {
            count[ibuf[tid]] = tid; }
    iuniq[tid] = _REL_ID_;
    __syncthreads();

    // exceed threshold
    if (tid > 0 && count[tid]-count[tid-1]>T) {
        iuniq[tid-1] = idx[count[tid]-1]; }
    __syncthreads();

    bitonic_sort(iuniq, ord_uniq);

    int temp = ord_uniq[ibuf[tid]];
    if (iuniq[temp] < _REL_ID_){
        ibuf[tid] = temp;
    }else{
        ibuf[tid] = idx[tid] + C;
    }
    if (iuniq[tid] < _REL_ID_ && iuniq[tid+1] == _REL_ID_){
        uniq_cnt[blockIdx.x] = tid+1;
    }
    buf_idx[blockIdx.x * C + tid] = ibuf[ord[tid]];
    uniq_idx[blockIdx.x * C + tid] = iuniq[tid];
}